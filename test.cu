#include "hip/hip_runtime.h"
#include "test.cuh"

__global__ void cuda_lanch_add(float *ptr, const int len)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < len; i = i + stride)
    {
        if(i < len){
            ptr[i] = ptr[i] * 2;
        }
    }
}


void add(float *ptr, const int len)
{
    dim3 blockSize(256);
    dim3 gridSize(2);
    cuda_lanch_add<<<gridSize, blockSize>>>(ptr, len);
}